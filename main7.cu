
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

#define uint unsigned int
#define ll long long
#define vl vector<ll>
#define MAX_VAL ((1LL << 32) - 1LL)

__global__ void matrixMul(int *vA, int *vvA, int *vB, int *vvB, uint *vC, int n, int m)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < (n / m) && col < (n / m))
    {
        for (int k = 0; k < (n / m); ++k)
        {
            //blocks to be multiplied are (row,k) and (k,col)
            if(vA[row*(n/m) + k] != 0 && vB[k*(n/m) + col]!=0){
            for(int ii=0;ii<m;++ii){
                for(int kk=0;kk<m;++kk){
                    for(int jj=0;jj<m;++jj){
                        vC[(row*(n/m) + col)*m*m+ ii*m+jj] += vvA[vA[row*(n/m) + k] -1 + ii*m+kk]  * vvB[vB[k*(n/m) + col]-1 + kk*m+jj];
                    }
                }
            }
            }
        }
    }
}

int main(int argc, char **argv)
{
    unsigned char bytes[4];
    unsigned char bytes1[2];

    int n = 0, m = 0, k1 = 0, starti, startj;

    FILE *fp = fopen(argv[1], "rb");

    fread(bytes, 4, 1, fp);
    n = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    m = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    k1 = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));

    int *vA;
    int *vvA;

    hipMallocManaged(&vA, (n / m) * (n / m) * sizeof(int));
    hipMallocManaged(&vvA, k1 * m * m * sizeof(int));

    int cnt = 0;

    while (cnt < k1)
    {
        fread(bytes, 4, 1, fp);
        starti = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
        fread(bytes, 4, 1, fp);
        startj = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));

        for (int ti = 0; ti < m; ++ti)
        {
            for (int tj = 0; tj < m; ++tj)
            {
                fread(bytes1, 2, 1, fp);
                vvA[cnt * m * m + ti * m + tj] = (bytes1[0] | (bytes1[1] << 8));
            }
        }
        vA[starti * (n / m) + startj] = cnt * m*m+1;
        cnt++;
    }
    // cout << "m1\n";
    // for (int i = 0; i < (n/m); ++i)
    // {
    //     for (int j = 0; j < (n/m); ++j)
    //     {
    //         cout << vA[i * (n/m) + j] << ' ';
    //     }
    //     cout << '\n';
    // }
    // for (int i=0;i<k1*m*m;++i) cout<<vvA[i]<<' ';
    // cout<<'\n';
    fp = fopen(argv[2], "rb");

    fread(bytes, 4, 1, fp);
    n = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    m = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    int k2 = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));

    int *vB;
    int *vvB;

    hipMallocManaged(&vB, (n / m) * (n / m) * sizeof(int));
    hipMallocManaged(&vvB, k2 * m * m * sizeof(int));

    cnt = 0;

    while (cnt < k2)
    {
        fread(bytes, 4, 1, fp);
        starti = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
        fread(bytes, 4, 1, fp);
        startj = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));

        for (int ti = 0; ti < m; ++ti)
        {
            for (int tj = 0; tj < m; ++tj)
            {
                fread(bytes1, 2, 1, fp);
                vvB[cnt * m * m + ti * m + tj] = (bytes1[0] | (bytes1[1] << 8));
            }
        }
        vB[starti * (n / m) + startj] = cnt * m*m+1;
        cnt++;
    }

    uint *vC;

    //cudaError_t err1;

    hipMallocManaged(&vC, n * n * sizeof(uint)); //(1<<30)

    //err1 = cudaPeekAtLastError();
    //cudaDeviceSynchronize();
    //printf("Got CUDA error ... %s \n", cudaGetErrorString(err1));

    int threads = 32;
    int blocks = ((n/m) + threads - 1) / threads;
    //cout<<blocks<<'\n';
    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    matrixMul<<<BLOCKS, THREADS>>>(vA,vvA,vB,vvB,vC,n,m);
    // cout << "m2\n";
    // for (int i = 0; i < n; ++i)
    // {
    //     for (int j = 0; j < n; ++j)
    //     {
    //         cout << b[i * n + j] << ' ';
    //     }
    //     cout << '\n';
    // }

    //err1 = cudaPeekAtLastError();
    hipDeviceSynchronize();
    //printf("Got CUDA error ... %s \n", cudaGetErrorString(err1));


    int total = 0;
    vector<int> indices;

    for (int i  = 0; i< n*n; i+= m*m){
        int f=0;
        for(int j=i;j<m*m+i;++j){
            if(vC[j]!=0){
                f=1;break;
            }
        }
        if(f==1) {total++;indices.push_back(i);}
    }
    // cout << "m3\n";
    // for (int i = 0; i < n*n; i+=m*m)
    // {
    //     for (int j = i; j < m*m + i; ++j)
    //     {
    //         cout << vC[j] << ' ';
    //     }
    //     cout << '\n';
    // }

    ofstream file(argv[3], ios::binary);
    file.write((char *)&n, 4);
    file.write((char *)&m, 4);
    file.write((char *)&total, 4);
    //cout<<total<<'\n';
    for (int i = 0; i < indices.size(); ++i)
    {
        int i1 = indices[i]/(m*m);
        int r = i1/(n/m);
        int c=i1%(n/m);
        file.write((char *)&(r), 4);
        file.write((char *)&(c), 4);
        //cout<<r<<' '<<c<<'\n';
        // cout<<ans[i].row<<' '<<ans[i].col<<'\n';
        for (int k = i1 * m *m; k < i1*m*m+m*m; ++k)
        {
            file.write((char *)(&vC[k]), 4);
            
        }
    }
    return 0;

}