
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    int a[3] = {1, 2, 3};
    int b[3] = {4, 5, 6};
    int c[3];

    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, 3 * sizeof(int));
    hipMalloc((void**)&dev_b, 3 * sizeof(int));
    hipMalloc((void**)&dev_c, 3 * sizeof(int));

    hipMemcpy(dev_a, a, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, 3 * sizeof(int), hipMemcpyHostToDevice);

    add<<<1,3>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, 3 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 3; i++)
        printf("%d ", c[i]);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
