
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

#define ll long long
#define vl vector<ll>
#define MAX_VAL ((1LL << 32) - 1LL)

__global__ void matrixMul(int *a, int *b, ll *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    {
        ll tmp = 0;
        for (int i = 0; i < n; i++)
        {
            tmp = min(MAX_VAL,tmp +  (ll)a[row * n + i] * (ll)b[i * n + col]);
        }
        c[row * n + col] = tmp;
    }
}

int main(int argc, char **argv)
{
    unsigned char bytes[4];
    unsigned char bytes1[2];

    int n = 0, m = 0, k1 = 0, starti, startj;

    FILE *fp = fopen(argv[1], "rb");

    fread(bytes, 4, 1, fp);
    n = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    m = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    k1 = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));

    int *a;

    hipMallocManaged(&a, n * n); //(1<<30)
    int cnt = 0;

    while (cnt < k1)
    {
        fread(bytes, 4, 1, fp);
        starti = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
        fread(bytes, 4, 1, fp);
        startj = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
        // cout<<starti<<' '<<startj<<'\n';
        for (int ti = 0; ti < m; ++ti)
        {
            for (int tj = 0; tj < m; ++tj)
            {
                fread(bytes1, 2, 1, fp);
                // cout<<(bytes[0] | (bytes[1] << 8))<<" same as ";
                a[(starti * m + ti) * n + (startj * m + tj)] = (bytes1[0] | (bytes1[1] << 8));
                // cout<<a[((starti)*m+ti)*n+(startj*m+tj)]<<' ';
            }
            //   cout<<'\n';
        }
        // cout<<'\n';
        cnt++;
    }
    cout << "m1\n";
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cout << a[i * n + j] << ' ';
        }
        cout << '\n';
    }
    fp = fopen(argv[2], "rb");

    fread(bytes, 4, 1, fp);
    n = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    m = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
    fread(bytes, 4, 1, fp);
    int k2 = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));

    int *b;
    ll *c;

    hipMallocManaged(&b, n * n); //(1<<30)
    hipMallocManaged(&c, n * n); //(1<<30)
    cnt = 0;

    while (cnt < k2)
    {
        fread(bytes, 4, 1, fp);
        starti = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
        fread(bytes, 4, 1, fp);
        startj = (bytes[0] | (bytes[1] << 8) | (bytes[2] << 16) | (bytes[3] << 24));
        // cout<<starti<<' '<<startj<<'\n';
        for (int ti = 0; ti < m; ++ti)
        {
            for (int tj = 0; tj < m; ++tj)
            {
                fread(bytes1, 2, 1, fp);
                b[(starti * m + ti) * n + (startj * m + tj)] = (bytes1[0] | (bytes1[1] << 8));
                // cout<<b[((starti)*m+ti)*n+(startj*m+tj)]<<' ';
            }
            // cout<<'\n';
        }
        // cout<<'\n';
        cnt++;
    }
    cout << "m2\n";
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cout << b[i * n + j] << ' ';
        }
        cout << '\n';
    }

    int threads = 32;
    int blocks = (n + threads - 1) / threads;

    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    matrixMul<<<BLOCKS, THREADS>>>(a, b, c, n);
    hipDeviceSynchronize();

    int total = 0;
    vector<pair<int, int>> indices;

    for (int i = 0; i < (n / m); ++i)
    {
        for (int j = 0; j < (n / m); ++j)
        {
            int f = 0;
            for (int k = i * m; k < i * m + m; ++k)
            {
                for (int l = j * m; l < j * m + m; ++l)
                {
                    if (c[k * n + l] != 0)
                    {
                        f = 1;
                        break;
                    }
                }
                if (f == 1)
                    break;
            }
            if (f == 1)
            {
                total++;
                indices.push_back(make_pair(i, j));
                // cout<<i<<' '<<j<<'\n';
            }
        }
    }
    cout << "m3\n";
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cout << c[i * n + j] << ' ';
        }
        cout << '\n';
    }

    ofstream file(argv[3], ios::binary);
    file.write((char *)&n, 4);
    file.write((char *)&m, 4);
    file.write((char *)&total, 4);
    for (int i = 0; i < indices.size(); ++i)
    {
        int i1 = indices[i].first;
        int i2 = indices[i].second;
        file.write((char *)&i1, 4);
        file.write((char *)&i2, 4);
        // cout<<ans[i].row<<' '<<ans[i].col<<'\n';
        for (int k = i1 * m; k < i1 * m + m; ++k)
        {
            for (int l = i2 * m; l < i2 * m + m; ++l)
            {
                file.write((char *)(&c[k * n + l]), 4);
            }
        }
    }
    return 0;
}
