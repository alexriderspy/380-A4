#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>

using namespace std;

#define ll long long
#define vl vector<ll>
#define MAX_VAL ((1LL<<32)-1LL)

struct block{
    int row,col;
    vl matrix;
};

bool cmp(const block& b1, const block& b2){
    if (b1.row == b2.row){
        return b1.col < b2.col;
    }
    return b1.row < b2.row;
}

__global__ void matrixMul(){

}

int main(int argc, char ** argv){
    unsigned char bytes[4];
    unsigned char bytes1[2];

    int n = 0, m = 0, k1 = 0, starti, startj;

    FILE *fp=fopen(argv[1],"rb");
    
    fread(bytes,4,1,fp);
    n  = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));
    fread(bytes,4,1,fp);
    m  = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));
    fread(bytes,4,1,fp);
    k1  = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));
    
    
    int blks = (n/m);

    int *v1_ind;
    int *v1_val;
    hipMallocManaged(&v1_ind, blks*blks); //(1<<30)
    hipMallocManaged(&v1_val, k1*8*(m*m)); // row, col, m*m 

    int step = 8*m*m;
    int cnt=0;
    // map<pair<int,int>,vl>vv2_1;
    // vector<block>vv_1(k1);
    
    int i=0,j=0;
    while(cnt < k1){
        fread(bytes, 4,1,fp);
        starti = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));
        fread(bytes, 4,1,fp);
        startj = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));

        v1_ind[starti*blks+startj] = cnt;
        int s = cnt;
        v1_val[s++] = starti;
        v1_val[s++] = startj;

        for (int ti=0;ti<m;++ti){
            for(int tj=0;tj<m;++tj){
                fread(bytes1,2,1,fp);
                v1_val[s++] = (bytes1[0] | (bytes1[1]<<8)); 
            }
        }
        //vv2_1[{starti, startj}] = mat;
        //vv_1[cnt] = {starti, startj, mat};
        cnt++;
    }

    fp=fopen(argv[2],"rb");
    
    fread(bytes,4,1,fp);
    n  = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));
    fread(bytes,4,1,fp);
    m  = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));
    fread(bytes,4,1,fp);
    int k2  = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));

    int *v2_ind;
    int *v2_val;
    hipMallocManaged(&v2_ind, blks*blks); //(1<<30)
    hipMallocManaged(&v2_val, k2*8*(m*m)); // row, col, m*m 

    step = 8*m*m;
    cnt=0;
    // map<pair<int,int>,vl>vv2_1;
    // vector<block>vv_1(k1);
    
    i=0,j=0;
    while(cnt < k1){
        fread(bytes, 4,1,fp);
        starti = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));
        fread(bytes, 4,1,fp);
        startj = (bytes[0] | (bytes[1]<<8) | (bytes[2]<<16) | (bytes[3]<<24));

        v2_ind[starti*blks+startj] = cnt;
        int s = cnt;
        v2_val[s++] = starti;
        v2_val[s++] = startj;

        for (int ti=0;ti<m;++ti){
            for(int tj=0;tj<m;++tj){
                fread(bytes1,2,1,fp);
                v2_val[s++] = (bytes1[0] | (bytes1[1]<<8)); 
            }
        }
        //vv2_1[{starti, startj}] = mat;
        //vv_1[cnt] = {starti, startj, mat};
        cnt++;
    }

    // cout<<'\n';
    map<pair<int,int>, vl> mp;

    int total_mp = 0;
    //do -1 before using 
    for(i=0;i<k1*step;i+=step){
        for(j=0;j<k2*step;j+=step){
            int r, c, index;
            if(v1_val[i+1] == v2_val[j]){
                r=v1_val[i], c=v2_val[j+1], index = v1_val[i+1];

                if (mp.find({r,c})!=mp.end()){
                    mp[{r,c}].push_back(index);
                    total_mp+=3;
                }else{
                    mp[{r,c}] = {index};
                    total_mp++;
                }
            }
        }
    }

    int mpsize = mp.size();

    int *f_ind;
    int *f_val;

    hipMallocManaged(&f_ind, mpsize+1); //+1 to have the last entry
    hipMallocManaged(&f_val, total_mp);

    //vector<block> finals(mpsize);
    int index_val=0, index_ind =0;

    for(auto &x:mp){
        f_ind[index_ind++] = index_val;
        f_val[index_val++] = x.first.first;
        f_val[index_val++] = x.first.second;
        for (int i=0;i<x.second.size();++i)
            f_val[index_val++] = x.second[i];
    }

    f_ind[index_ind] = index_val;

    vector<block> ans(mpsize);

    int total = 0;

    vector<int>is0(mpsize);

    int THREADS = 1000;
    int BLOCKS = (mpsize + THREADS -1)/THREADS;


    matrixMul<<<BLOCKS, THREADS>>>();
    
    __global__ void matrixMul();{

        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        if (tid < mpsize){
            vl out(m*m);
            
            int r=f_val[f_ind[tid]], c=f_val[f_ind[tid]+1];
            for(int j=f_ind[tid]+2;j<f_ind[tid+1];++j){

                int index = f_val[j];
                vl inn(m*m);
                vl fm,sm;
                

                //multiply (r,index) * (index, c)
                fm = vv2_1[{finals[i].row,finals[i].matrix[j]}];
                sm = vv2_2[{finals[i].matrix[j],finals[i].col}];
                
                for(int ii=0;ii<m;ii++)
                    for(int kk=0;kk<m;kk++)
                        for(int jj=0;jj<m;jj++)
                            inn[ii*m + jj]=min(MAX_VAL,inn[ii*m+jj]+min(MAX_VAL,fm[ii*m+kk]*sm[kk*m+jj]));

                for (int ii=0;ii<m*m;++ii){
                    out[ii] = min(MAX_VAL,out[ii]+inn[ii]);
                }
            }
            ans[i] = {finals[i].row, finals[i].col,out};
        }
        
    }
    // //{
    // for(int i=0;i<mpsize;++i){
    //     vl out(m*m);
        
    //     for(int j=0;j<finals[i].matrix.size();++j){
    //         vl inn(m*m);
    //         vl fm,sm;

    //         fm = vv2_1[{finals[i].row,finals[i].matrix[j]}];
    //         sm = vv2_2[{finals[i].matrix[j],finals[i].col}];
            
    //         for(int ii=0;ii<m;ii++)
    //             for(int kk=0;kk<m;kk++)
    //                 for(int jj=0;jj<m;jj++)
    //                     inn[ii*m + jj]=min(MAX_VAL,inn[ii*m+jj]+min(MAX_VAL,fm[ii*m+kk]*sm[kk*m+jj]));

    //         for (int ii=0;ii<m*m;++ii){
    //             out[ii] = min(MAX_VAL,out[ii]+inn[ii]);
    //         }
    //     }
    //     ans[i] = {finals[i].row, finals[i].col,out};
    // }
    // //}

    // for(i=0;i<mpsize;++i){
    //     if (count(ans[i].matrix.begin(),ans[i].matrix.end(),0)!=m*m){
    //         total ++;
    //     }else{
    //         is0[i]=1;
    //     }
    // }
    
    // thrust::sort(thrust::host, ans.begin(), ans.end(), cmp);
    // //sort(ans.begin(),ans.end(), cmp);
    // ofstream file(argv[3],ios::binary);
    // file.write((char*)&n,4);
    // file.write((char*)&m,4);
    // file.write((char*)&total,4);
    // for (i=0;i<mpsize;++i){
    //     if(is0[i]!=1){
    //         file.write((char*)&ans[i].row,4);
    //         file.write((char*)&ans[i].col,4);
    //         //cout<<ans[i].row<<' '<<ans[i].col<<'\n';
    //         for (j=0;j<m*m;++j){
    //             file.write((char*)(&(ans[i].matrix[j])),4);
    //             //cout<<ans[i].matrix[j]<<' ';
    //         }
    //         //cout<<'\n';
    //     }
    // }
    return 0;
}